#include "hip/hip_runtime.h"
// Copyright (C) 2021 THL A29 Limited, a Tencent company.  All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software distributed under the License
// is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express
// or implied. See the License for the specific language governing permissions and limitations under
// the License.
//
// ╔════════════════════════════════════════════════════════════════════════════════════════╗
// ║──█████████╗───███████╗───████████╗───██╗──────██╗───███████╗───████████╗───████████╗───║
// ║──██╔══════╝──██╔════██╗──██╔════██╗──██║──────██║──██╔════██╗──██╔════██╗──██╔════██╗──║
// ║──████████╗───██║────██║──████████╔╝──██║──█╗──██║──█████████║──████████╔╝──██║────██║──║
// ║──██╔═════╝───██║────██║──██╔════██╗──██║█████╗██║──██╔════██║──██╔════██╗──██║────██║──║
// ║──██║─────────╚███████╔╝──██║────██║──╚████╔████╔╝──██║────██║──██║────██║──████████╔╝──║
// ║──╚═╝──────────╚══════╝───╚═╝────╚═╝───╚═══╝╚═══╝───╚═╝────╚═╝──╚═╝────╚═╝──╚═══════╝───║
// ╚════════════════════════════════════════════════════════════════════════════════════════╝
//
// Authors: Aster JIAN (asterjian@qq.com)
//          Yzx (yzxyzxyzx777@outlook.com)
//          Ao LI (346950981@qq.com)
//          Paul LU (lujq96@gmail.com)

#include "trt_engine/trt_network_crt/plugins/cast_plugin/cast_plugin.h"

#include <hip/hip_fp16.h>

FWD_TRT_NAMESPACE_BEGIN
template <typename in_t, typename out_t>
__global__ void CastKernel(const in_t* input, out_t* out, size_t size) {
  const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < size) {
    out[idx] = static_cast<out_t>(input[idx]);
  }
}

template <typename in_T, typename out_T>
void Cast(const in_T* input, out_T* output, size_t size) {
  const int blockDim = 1024;
  const int gridDim = static_cast<int>((size + blockDim - 1) / blockDim);

  CastKernel<in_T, out_T>
      <<<gridDim, blockDim>>>(static_cast<const in_T*>(input), static_cast<out_T*>(output), size);
}

template void Cast<half, float>(const half* input, float* output, size_t size);
template void Cast<int, float>(const int* input, float* output, size_t size);
template void Cast<int8_t, float>(const int8_t* input, float* output, size_t size);
template void Cast<bool, float>(const bool* input, float* output, size_t size);

FWD_TRT_NAMESPACE_END
